/* MIT License
 *
 * Copyright (c) 2024 Maximilian Behr
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

#include <hipsolver.h>

#include <chrono>
#include <cstdio>
#include <cstdlib>

#include "cugje.h"

int main(int argc, char **argv) {
    /*-----------------------------------------------------------------------------
     * variables
     *-----------------------------------------------------------------------------*/
    int ret = 0;                              // return value
    int n = 1000;                             // size of the input matrix n-by-n
    int m = n;                                // size of the right-hand side matrix n-by-m
    double *A, *RHS;                          // input matrix and right-hand side matrix
    double *dA, *dRHS, *dA2, *dRHS2;          // device matrices
    double *dbuffer = NULL, *hbuffer = NULL;  // buffer for cusolver
    int64_t *d_ipiv = NULL;                   // pivoting sequence
    int *d_info = NULL;                       // error code

    /*-----------------------------------------------------------------------------
     * parse command line arguments
     *-----------------------------------------------------------------------------*/
    if (argc > 1) {
        n = atoi(argv[1]);
    }
    if (argc > 2) {
        m = atoi(argv[2]);
    }
    if (argc > 3) {
        printf("Usage: %s [n] [m]\n", argv[0]);
        return 1;
    }

    /*-----------------------------------------------------------------------------
     * allocate
     *-----------------------------------------------------------------------------*/
    hipHostMalloc((void **)&A, sizeof(*A) * n * n);
    hipHostMalloc((void **)&RHS, sizeof(*RHS) * n * m);
    hipMalloc((void **)&dA, sizeof(*dA) * n * n);
    hipMalloc((void **)&dRHS, sizeof(*dRHS) * n * m);
    hipMalloc((void **)&dA2, sizeof(*dA2) * n * n);
    hipMalloc((void **)&dRHS2, sizeof(*dRHS2) * n * m);
    hipMalloc((void **)&d_ipiv, sizeof(*d_ipiv) * n);
    hipMalloc((void **)&d_info, sizeof(*d_info));

    /*-----------------------------------------------------------------------------
     * create a random matrix A and RHS
     *-----------------------------------------------------------------------------*/
    srand(0);
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            A[i + j * n] = (double)rand() / RAND_MAX;
        }
        for (int j = 0; j < m; ++j) {
            RHS[i + j * n] = (double)rand() / RAND_MAX;
        }
    }

    /*-----------------------------------------------------------------------------
     * copy data to the device
     *-----------------------------------------------------------------------------*/
    hipMemcpy(dA, A, n * n * sizeof(*A), hipMemcpyHostToDevice);
    hipMemcpy(dRHS, RHS, n * m * sizeof(*RHS), hipMemcpyHostToDevice);
    hipMemcpy(dA2, A, n * n * sizeof(*A), hipMemcpyHostToDevice);
    hipMemcpy(dRHS2, RHS, n * m * sizeof(*RHS), hipMemcpyHostToDevice);

    /*-----------------------------------------------------------------------------
     * measure the time and perform Gauss-Jordan Elimination on the device
     *-----------------------------------------------------------------------------*/
    double wtime_cugjed = 0.0;
    {
        auto start = std::chrono::high_resolution_clock::now();
        cugjed(n, m, dA, dRHS);
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed = end - start;
        wtime_cugjed = elapsed.count();
    }

    /*-----------------------------------------------------------------------------
     * measaure the time and solve the linear system with cusolver
     *-----------------------------------------------------------------------------*/
    hipsolverHandle_t cusolverH;
    hipsolverDnCreate(&cusolverH);

    hipsolverDnParams_t params;
    hipsolverDnCreateParams(&params);
    hipsolverDnSetAdvOptions(params, HIPSOLVERDN_GETRF, HIPSOLVER_ALG_0);

    size_t lworkdevice = 0, lworkhost = 0;
    hipsolverDnXgetrf_bufferSize(cusolverH, params, n, n, HIP_R_64F, dA2, n, HIP_R_64F, &lworkdevice, &lworkhost);

    if (lworkdevice > 0) {
        hipMalloc((void **)&dbuffer, lworkdevice);
    }

    if (lworkhost > 0) {
        hipHostMalloc((void **)&hbuffer, lworkhost);
    }

    double wtime_cusolver = 0.0;
    {
        auto start = std::chrono::high_resolution_clock::now();
        hipsolverDnXgetrf(cusolverH, params, n, n, HIP_R_64F, dA2, n, d_ipiv, HIP_R_64F, dbuffer, lworkdevice, hbuffer, lworkhost, d_info);
        hipsolverDnXgetrs(cusolverH, params, HIPBLAS_OP_N, n, m, HIP_R_64F, dA2, n, d_ipiv, HIP_R_64F, dRHS2, n, d_info);
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed = end - start;
        wtime_cusolver = elapsed.count();
    }

    /*-----------------------------------------------------------------------------
     * print results
     *-----------------------------------------------------------------------------*/
    printf("%d,%d,%e,%e\n", n, m, wtime_cugjed, wtime_cusolver);

    /*-----------------------------------------------------------------------------
     * destroy the cusolver handle
     *-----------------------------------------------------------------------------*/
    hipsolverDnDestroy(cusolverH);

    /*-----------------------------------------------------------------------------
     * clear memory
     *-----------------------------------------------------------------------------*/
    hipHostFree(A);
    hipHostFree(RHS);
    hipFree(dA);
    hipFree(dRHS);
    hipFree(dA2);
    hipFree(dRHS2);
    hipFree(dbuffer);
    hipHostFree(hbuffer);
    hipFree(d_ipiv);
    hipFree(d_info);

    return ret;
}
